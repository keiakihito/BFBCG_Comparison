// includes, system
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/*Using updated (v2) interfaces to cublas*/
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include <hipsolver.h>
#include<sys/time.h>


//Utilities
#include "../includes/helper_debug.h"
// helper function CUDA error checking and initialization
#include "../includes/hip/hip_runtime_api.h"  
#include "../includes/helper_functions.h"
#include "../includes/cusolver_utils.h"
#include "../includes/CSRMatrix.h"


#define CHECK(call){ \
    const hipError_t cuda_ret = call; \
    if(cuda_ret != hipSuccess){ \
        printf("Error: %s:%d,  ", __FILE__, __LINE__ );\
        printf("code: %d, reason: %s \n", cuda_ret, hipGetErrorString(cuda_ret));\
        exit(-1); \
    }\
}


void sparseDenseMultiplyTest_Case1();
void sparseDenseMultiplyTest_Case2();
void sparseDenseMultiplyTest_Case3();
void sparseDenseMultiplyTest_Case4();
void sparseDenseMultiplyTest_Case5();

int main(int arg, char** argv)
{
    
    printf("\n\n= = = =sparseDenseMultiplyTest.cu= = = = \n\n");
    
    // printf("\n\n🔍🔍🔍 Test Case 1 🔍🔍🔍\n\n");
    // sparseDenseMultiplyTest_Case1();

    // printf("\n\n🔍🔍🔍 Test Case 2 🔍🔍🔍\n\n");
    // sparseDenseMultiplyTest_Case2();

    // printf("\n\n🔍🔍🔍 Test Case 3 🔍🔍🔍\n\n");
    // sparseDenseMultiplyTest_Case3();

    // printf("\n\n🔍🔍🔍 Test Case 4 🔍🔍🔍\n\n");
    // sparseDenseMultiplyTest_Case4();

    // printf("\n\n🔍🔍🔍 Test Case 5 🔍🔍🔍\n\n");
    // sparseDenseMultiplyTest_Case5();

    printf("\n\n= = = = end of sparseDenseMultiplyTest = = = =\n\n");

    return 0;
} // end of main




void sparseDenseMultiplyTest_Case1()
{

} // end of sparseDenseMultiplyTest_Case1




void sparseDenseMultiplyTest_Case2()
{

} // end of sparseDenseMultiplyTest_Case2




void sparseDenseMultiplyTest_Case3()
{

} // end of sparseDenseMultiplyTest_Case1




void sparseDenseMultiplyTest_Case4()
{

} // end of sparseDenseMultiplyTest_Case1




void sparseDenseMultiplyTest_Case5()
{

} // end of sparseDenseMultiplyTest_Case1