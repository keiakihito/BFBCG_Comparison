// includes, system
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/*Using updated (v2) interfaces to cublas*/
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include <hipsolver.h>
#include<sys/time.h>


//Utilities
#include "../includes/helper_debug.h"
// helper function CUDA error checking and initialization
#include "../includes/hip/hip_runtime_api.h"  
#include "../includes/helper_functions.h"
#include "../includes/cusolver_utils.h"


#define CHECK(call){ \
    const hipError_t cuda_ret = call; \
    if(cuda_ret != hipSuccess){ \
        printf("Error: %s:%d,  ", __FILE__, __LINE__ );\
        printf("code: %d, reason: %s \n", cuda_ret, hipGetErrorString(cuda_ret));\
        exit(-1); \
    }\
}


void sparseMulTest_Case1();
void sparseMulTest_Case2();
void sparseMulTest_Case3();
void sparseMulTest_Case4();
void sparseMulTest_Case5();

int main(int arg, char** argv)
{
    printf("\n\nHello World from sparseMulTest.cu\n\n");

    sparseMulTest_Case1();
    sparseMulTest_Case2();
    sparseMulTest_Case3();
    sparseMulTest_Case4();
    sparseMulTest_Case5();

    return 0;
} // end of main


void sparseMulTest_Case1()
{

}


void sparseMulTest_Case2()
{
    
}


void sparseMulTest_Case3()
{
    
}


void sparseMulTest_Case4()
{
    
}


void sparseMulTest_Case5()
{
    
}