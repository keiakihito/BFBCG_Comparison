// includes, system
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/*Using updated (v2) interfaces to cublas*/
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include<sys/time.h>


//Utilities
#include "includes/helper_debug.h"
// helper function CUDA error checking and initialization
#include "includes/hip/hip_runtime_api.h"  
#include "includes/helper_functions.h"

#define CHECK(call){ \
    const hipError_t cuda_ret = call; \
    if(cuda_ret != hipSuccess){ \
        printf("Error: %s:%d,  ", __FILE__, __LINE__ );\
        printf("code: %d, reason: %s \n", cuda_ret, hipGetErrorString(cuda_ret));\
        exit(-1); \
    }\
}

//Bigger size matrix
#define N 5 //


// Define the dense matrixB
float denseMtxB[] = {
    0.1, 0.6, 1.1,
    0.2, 0.7, 1.2,
    0.3, 0.8, 1.3,
    0.4, 0.9, 1.4,
    0.5, 1.0, 1.5
};







// Time tracker for each iteration
double myCPUTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec/1.0e6);
}




int main(int argc, char** argv)
{   
    // double startTime, endTime;
    int row[] = {0, 2, 5, 8, 11, 13};
    int col[] = {0, 1, 0, 1, 2, 1, 2, 3, 2, 3, 4, 3, 4};
    float val[] = {10, 1, 1, 20, 1, 1, 30, 1, 1, 40, 1, 1, 50};

    //For sparse matrix A
    int numRows = 5;
    int numCols = 5;
    int nnz = 13;// Number of Non zero

    //For dense matrix B
    int numRows_B = 5;
    int numCols_B = 3;

    //(1) Allocate device memory
    int *row_d = NULL;
    int *col_d = NULL;
    float *val_d = NULL;

    float *dnsMtxB_d = NULL;
    float *dnsMtxAB_d = NULL;// Result

    CHECK(hipMalloc((void**)&row_d, (numRows+1) * sizeof(int)));
    CHECK(hipMalloc((void**)&col_d, numCols * sizeof(int)));
    CHECK(hipMalloc((void**)&val_d, nnz * sizeof(float)));

    CHECK(hipMalloc((void**)&dnsMtxB_d, numRows_B * numCols_B * sizeof(float)));
    CHECK(hipMalloc((void**)&dnsMtxAB_d, numRows * numCols_B * sizeof(float)));


    //(2) Copy value to device
    CHECK(hipMemcpy(row_d, row, (numRows+1) *sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(col_d, col, numCols * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(val_d, val, nnz * sizeof(int), hipMemcpyHostToDevice));

    CHECK(hipMemcpy(dnsMtxB_d, denseMtxB, numRows_B * numCols_B * sizeof(float), hipMemcpyHostToDevice));
    

    //(3) Create cuspare handle and descreptors
    hipsparseSpMatDescr_t mtxA_dscr;
    hipsparseDnMatDescr_t mtxB_dscr, mtxC_dscr;

    checkCudaErrors(hipsparseCreateCsr(&mtxA_dscr, numRows, numCols, nnz, row_d, col_d, val_d,HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    checkCudaErrors(hipsparseCreateDnMat(&mtxB_dscr, numRows_B, numCols_B, numRows_B, dnsMtxB_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    checkCudaErrors(hipsparseCreateDnMat(&mtxC_dscr, numRows, numCols_B, numRows, dnsMtxAB_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));

    //(4) Computer sparse-dense matrix multiplication



    return 0;
} // end of main

