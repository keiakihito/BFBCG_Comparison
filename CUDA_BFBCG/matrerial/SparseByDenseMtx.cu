// includes, system
#include<stdio.h>
#include<stdlib.h>
#include<string.h>

/*Using updated (v2) interfaces to cublas*/
#include<hipblas.h>
#include<hip/hip_runtime.h>
#include<hipsparse.h>
#include<sys/time.h>


//Utilities
#include "includes/helper_debug.h"
// helper function CUDA error checking and initialization
#include "includes/hip/hip_runtime_api.h"  
#include "includes/helper_functions.h"

#define CHECK(call){ \
    const hipError_t cuda_ret = call; \
    if(cuda_ret != hipSuccess){ \
        printf("Error: %s:%d,  ", __FILE__, __LINE__ );\
        printf("code: %d, reason: %s \n", cuda_ret, hipGetErrorString(cuda_ret));\
        exit(-1); \
    }\
}

//Bigger size matrix
#define N 5 //


// Define the dense matrixB
float denseMtxB[] = {
    0.1, 0.6, 1.1,
    0.2, 0.7, 1.2,
    0.3, 0.8, 1.3,
    0.4, 0.9, 1.4,
    0.5, 1.0, 1.5
};



// Time tracker for each iteration
double myCPUTimer()
{
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec/1.0e6);
}




int main(int argc, char** argv)
{   
    // double startTime, endTime;
    int row[] = {0, 2, 5, 8, 11, 13};
    int col[] = {0, 1, 0, 1, 2, 1, 2, 3, 2, 3, 4, 3, 4};
    float val[] = {10.0, 1.0, 1.0, 20.0, 1.0, 1.0, 30.0, 1.0, 1.0, 40.0, 1.0, 1.0, 50.0};

    //For sparse matrix A
    int numRows = 5;
    int numCols = 5;
    int nnz = 13;// Number of Non zero

    //For dense matrix B
    int numRows_B = 5;
    int numCols_B = 3;

    float alpha = 1.0;
    float beta = 0.0;

    //(1) Allocate device memory
    int *row_d = NULL;
    int *col_d = NULL;
    float *val_d = NULL;

    float *dnsMtxB_d = NULL;
    float *dnsMtxAB_h = NULL;// Result in host
    float *dnsMtxAB_d = NULL;// Result in device

    bool debug = false;

    CHECK(hipMalloc((void**)&row_d, (numRows+1) * sizeof(int)));
    CHECK(hipMalloc((void**)&col_d, nnz * sizeof(int)));
    CHECK(hipMalloc((void**)&val_d, nnz * sizeof(float)));
    

    CHECK(hipMalloc((void**)&dnsMtxB_d, numRows_B * numCols_B * sizeof(float)));
    CHECK(hipMalloc((void**)&dnsMtxAB_d, numRows * numCols_B * sizeof(float)));


    //(2) Copy value to device
    CHECK(hipMemcpy(row_d, row, (numRows+1) *sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(col_d, col, nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(val_d, val, nnz * sizeof(float), hipMemcpyHostToDevice));

    debug = false;
    if(debug){
        printf("\n\nrow vector \n");
        print_vector(row_d, numRows+1);
        printf("\n\ncol vector \n");
        print_vector(col_d, nnz);
        printf("\n\nval vector \n");
        print_vector(val_d, nnz);
    }
    debug = false;


    CHECK(hipMemcpy(dnsMtxB_d, denseMtxB, numRows_B * numCols_B * sizeof(float), hipMemcpyHostToDevice));
    // print_mtx_d(dnsMtxB_d, numRows_B, numCols_B);   
    // print_mtx_d(dnsMtxAB_d, numRows, numCols_B);    

    //(3) Create cuspare handle and descreptors
    hipsparseSpMatDescr_t mtxA_dscr;
    hipsparseDnMatDescr_t mtxB_dscr, mtxC_dscr;

    hipsparseHandle_t cusparseHandle;
    hipsparseCreate(&cusparseHandle);

    
    checkCudaErrors(hipsparseCreateCsr(&mtxA_dscr, numRows, numCols, nnz, row_d, col_d, val_d,HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    //Note Given the marix is row major order (HIPSPARSE_ORDER_ROW), the leading dimension is number of column.
    checkCudaErrors(hipsparseCreateDnMat(&mtxB_dscr, numRows_B, numCols_B, numCols_B, dnsMtxB_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    checkCudaErrors(hipsparseCreateDnMat(&mtxC_dscr, numRows, numCols_B, numCols_B, dnsMtxAB_d, HIP_R_32F, HIPSPARSE_ORDER_ROW));
    
    debug = false;
    if(debug){
        printf("\n\nrow vector \n");
        print_vector(row_d, numRows+1);
        printf("\n\ncol vector \n");
        print_vector(col_d, nnz);
        printf("\n\nval vector \n");
        print_vector(val_d, nnz);

        printf("\n\ndnsMtxB\n");
        print_mtx_d(dnsMtxB_d, numRows_B, numCols_B);
        printf("\n\ndnsMtxAB\n");
        print_mtx_d(dnsMtxAB_d, numRows, numCols_B);   
    }
    debug = false;

 

    //(4) Computer sparse-dense matrix multiplication

    //Need to allocate buffer for hipsparseSpMM
    size_t bufferSize = 0;
    void* dBuffer = NULL;
    checkCudaErrors(hipsparseSpMM_bufferSize(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, mtxA_dscr, mtxB_dscr, &beta, mtxC_dscr, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize));

    CHECK(hipMalloc(&dBuffer, bufferSize));

    //Perform sparse * dense matrix operaroin
    checkCudaErrors(hipsparseSpMM(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
    &alpha, mtxA_dscr, mtxB_dscr, &beta, mtxC_dscr, HIP_R_32F, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));


  

    // (5) Copy back the result to host
    dnsMtxAB_h = (float*)malloc(sizeof(float)* numRows * numCols_B);
    CHECK(hipMemcpy(dnsMtxAB_h, dnsMtxAB_d, sizeof(float)*(numRows * numCols_B), hipMemcpyDeviceToHost));

    printf("\n\n~~Check sprMtxA * dnsMtxB~~\n");
    print_mtx_h(dnsMtxAB_h, numRows, numCols_B);

    //(6) Free pointers
    checkCudaErrors(hipsparseDestroySpMat(mtxA_dscr));
    checkCudaErrors(hipsparseDestroyDnMat(mtxB_dscr));
    checkCudaErrors(hipsparseDestroyDnMat(mtxC_dscr));
    checkCudaErrors(hipsparseDestroy(cusparseHandle));

    CHECK(hipFree(dBuffer));
    CHECK(hipFree(row_d));
    CHECK(hipFree(col_d));
    CHECK(hipFree(val_d));
    CHECK(hipFree(dnsMtxB_d));
    CHECK(hipFree(dnsMtxAB_d));
    
    free(dnsMtxAB_h);

    return 0;
} // end of main

